#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../library/mycuda.h"
#include <chrono>


using namespace std;

// a = b + c
__global__
void add(int *a, int *b, int *c, size_t N)
{

    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if ( id < N )
        a[id] = b[id] + c[id];
    
}

int main()
{

    size_t N = 800000;

    // host variables

    vector<int> a(N, 0);
    vector<int> b(N, 1);
    vector<int> c(N, 2);
    vector<int> d(N, 3);

    // block and grid dimensions
    dim3 blockDim;
    dim3 gridDim;

    calculateDimensions(N, blockDim, gridDim);

    // device pointers

    int* d_a;
    int* d_b;
    int* d_c;

    // allocate and copy to device

    CUDA_CALL( hipMalloc((void**)&d_a, sizeof(int) * N) );
    CUDA_CALL( hipMalloc((void**)&d_b, sizeof(int) * N) );
    CUDA_CALL( hipMalloc((void**)&d_c, sizeof(int) * N) );

    CUDA_CALL( hipMemcpy(d_a, &a[0], sizeof(int) * N, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(d_b, &b[0], sizeof(int) * N, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(d_c, &c[0], sizeof(int) * N, hipMemcpyHostToDevice) );
    
    


    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    
    
    
    
    // run kernel
    
    add<<<gridDim, blockDim>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
 
    
    std::chrono::steady_clock::time_point end= std::chrono::steady_clock::now();
    
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() <<std::endl;
    
    
    
    CUDA_CALL( hipMemcpy(&a[0], d_a, sizeof(int) * N, hipMemcpyDeviceToHost) );
    

    // verify d_c

    int checker = 0;

    for(int i = 0 ; i < d.size() ; ++i)
    {
        if ( a[i] != d[i] )
            checker++;
    }


    if ( checker > 0 )
        cout << "GPU: fail" << endl;
    
    else
        cout << "GPU: pass" << endl;

    // deallocate memory in device

    CUDA_CALL( hipFree(d_a) );
    CUDA_CALL( hipFree(d_b) );
    CUDA_CALL( hipFree(d_c) );


    // compare with CPU

    vector<int> h_a(N, 0);
    vector<int> h_b(N, 1);
    vector<int> h_c(N, 2);


    begin = std::chrono::steady_clock::now();
    
    for(int i = 0 ; i < d.size() ; ++i)
    {
        h_a[i] = h_b[i] + h_c[i];
    }
    
    end= std::chrono::steady_clock::now();
    
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() <<std::endl;
 
    int h_checker = 0;

    for(int i = 0 ; i < d.size() ; ++i)
    {
        if ( a[i] != d[i] )
            h_checker++;
    }

    if ( h_checker > 0 )
        cout << "CPU: fail" << endl;
    
    else
        cout << "CPU: pass" << endl;



}